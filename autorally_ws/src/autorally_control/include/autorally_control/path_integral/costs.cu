#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
3* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file costs.cu
 * @author Grady Williams <gradyrw@gmail.com>
 * @date May 24, 2017
 * @copyright 2017 Georgia Institute of Technology
 * @brief MPPICosts class implementation
 ***********************************************/
#include "gpu_err_chk.h"
#include "debug_kernels.cuh"

#include <stdio.h>
#include <stdlib.h>

namespace autorally_control {

inline MPPICosts::MPPICosts(int width, int height, int depth )
{
  width_ = width;
  height_ = height;
  depth_ = depth;
  allocateTexMem();
  //Initialize memory for device cost param struct
  HANDLE_ERROR( hipMalloc((void**)&params_d_, sizeof(CostParams)) );
  debugging_ = false;
  debugging_costmap_ = false;
  initCostmap();
  initCostmap3D(); //ST: init 3d costmap
  initObstacles(); //ST

}

inline MPPICosts::MPPICosts(ros::NodeHandle nh)
{
  //Transform from world coordinates to normalized grid coordinates
  Eigen::Matrix3f R;
  Eigen::Array3f trs;
  HANDLE_ERROR( hipMalloc((void**)&params_d_, sizeof(CostParams)) ); //Initialize memory for device cost param struct
  //Get the map path
  std::string map_path = getRosParam<std::string>("map_path", nh);
  //track_costs_ = loadTrackData(map_path, R, trs); //R and trs passed by reference
  obstacles_costs_ = loadTrackData(map_path, R, trs);   //ST:changes are made inside this function
  updateTransform(R, trs);
  updateParams(nh);
  allocateTexMem();
  costmapToTexture();
  obstaclesToTexture();
  costmap3DToTexture();
  debugging_ = false;
  debugging_costmap_ = false;
}

inline void MPPICosts::allocateTexMem()
{
  //Allocate memory for the cuda array which is bound the costmap_tex_
  channelDesc_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  HANDLE_ERROR(hipMallocArray(&costmapArray_d_, &channelDesc_, width_, height_));
  channelDescObs_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);   //ST
  HANDLE_ERROR(hipMallocArray(&obstaclesArray_d_, &channelDescObs_, width_, height_)); //ST
  channelDescCost3D_ = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);   //ST
  hipExtent extent = make_hipExtent(width_, height_, depth_); //ST: 3D texture in number of elements
  HANDLE_ERROR(hipMalloc3DArray(&costmap3DArray_d_, &channelDescCost3D_, extent)); //ST


}

inline void MPPICosts::updateParams_dcfg(autorally_control::PathIntegralParamsConfig config)
{
  params_.desired_speed = (float)config.desired_speed;
  params_.speed_coeff = (float)config.speed_coefficient;
  params_.track_coeff = (float)config.track_coefficient;
  params_.max_slip_ang = (float)config.max_slip_angle;
  params_.slip_penalty = (float)config.slip_penalty;
  params_.crash_coeff = (float)config.crash_coefficient;
  params_.track_slop = (float)config.track_slop;
  params_.steering_coeff = (float)config.steering_coeff;
  params_.throttle_coeff = (float)config.throttle_coeff;

  //ST
  //printf("Costs: Got Params User Linear Speed : %f\n", (float)config.user_desired_linear_speed);
  params_.user_desired_linear_speed = (float) config.user_desired_linear_speed;
  params_.user_desired_angular_speed = (float)config.user_desired_angular_speed;
  params_.angular_speed_coeff = (float)config.angular_speed_coefficient;
  params_.linear_speed_coeff = (float)config.linear_speed_coefficient;
  params_.smoothness_coeff = (float)config.smoothness_coefficient;
  params_.goal_coeff = (float)config.goal_coefficient;
  //end

  paramsToDevice();
}

inline void MPPICosts::initCostmap()
{
  track_costs_ = std::vector<float4>(width_*height_);
  //Initialize costmap to zeros
  for (int i = 0; i < width_*height_; i++){
    track_costs_[i].x = 0;
    track_costs_[i].y = 0;
    track_costs_[i].z = 0;
    track_costs_[i].w = 0;
  }
}

inline void MPPICosts::initObstacles()
{
    //ST: init obstacle map
    obstacles_costs_ = std::vector<float4>(width_*height_);
    //Initialize obstacles to zeros
    for (int i = 0; i < width_*height_; i++){
        obstacles_costs_[i].x = 0;
        obstacles_costs_[i].y = 0;
        obstacles_costs_[i].z = 0;
        obstacles_costs_[i].w = 0;
    }
}

inline void MPPICosts::initCostmap3D()
{
    //ST: init costmap 3D with zeros
    costmap3D_costs_ = std::vector<float4>(width_*height_*depth_);
    for (int i = 0; i < width_*height_*depth_; i++){
        costmap3D_costs_[i].x = 0.5;
        costmap3D_costs_[i].y = 0.5;
        costmap3D_costs_[i].z = 0.5;
        costmap3D_costs_[i].w = 0.5;
    }
}


inline void MPPICosts::costmapToTexture(float* costmap, int channel)
{

    switch(channel){
    case 0:
      for (int i = 0; i < width_*height_; i++){
        track_costs_[i].x = costmap[i];
      }
      break;
    case 1:
      for (int i = 0; i < width_*height_; i++){
        track_costs_[i].y = costmap[i];
      }
      break;
    case 2:
      for (int i = 0; i < width_*height_; i++){
        track_costs_[i].z = costmap[i];
      }
      break;
    case 3:
      for (int i = 0; i < width_*height_; i++){
        track_costs_[i].w = costmap[i];
      }
      break;
  }
  costmapToTexture();
}

inline void MPPICosts::obstaclesToTexture(float* obstacles, int channel)
{
    //ST
    switch(channel){
        case 0:
            for (int i = 0; i < width_*height_; i++){
                obstacles_costs_[i].x = obstacles[i];
            }
            break;
        case 1:
            for (int i = 0; i < width_*height_; i++){
                obstacles_costs_[i].y = obstacles[i];
            }
            break;
        case 2:
            for (int i = 0; i < width_*height_; i++){
                obstacles_costs_[i].z = obstacles[i];
            }
            break;
        case 3:
            for (int i = 0; i < width_*height_; i++){
                obstacles_costs_[i].w = obstacles[i];
            }
            break;
    }
    obstaclesToTexture();
}

inline void MPPICosts::costmap3DToTexture(float* costmap3D, int channel)
{
    //ST
    switch(channel){
        case 0:
            for (int i = 0; i < width_*height_*depth_; i++){
                costmap3D_costs_[i].x = costmap3D[i];
            }
            break;
        case 1:
            for (int i = 0; i < width_*height_*depth_; i++){
                costmap3D_costs_[i].y = costmap3D[i];
            }
            break;
        case 2:
            for (int i = 0; i < width_*height_*depth_; i++){
                costmap3D_costs_[i].z = costmap3D[i];
            }
            break;
        case 3:
            for (int i = 0; i < width_*height_*depth_; i++){
                costmap3D_costs_[i].w = costmap3D[i];
            }
            break;
    }
    costmap3DToTexture();

}

inline void MPPICosts::costmapToTexture()
{
  //costmap_ = costmap;
  //Transfer CPU mem to GPU
  float4* costmap_ptr = track_costs_.data();
  HANDLE_ERROR(hipMemcpyToArray(costmapArray_d_, 0, 0, costmap_ptr, width_*height_*sizeof(float4), hipMemcpyHostToDevice));
  hipStreamSynchronize(stream_);

  //Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = costmapArray_d_;

  //Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  //Destroy current texture and create new texture object
  HANDLE_ERROR(hipDestroyTextureObject(costmap_tex_));
  HANDLE_ERROR(hipCreateTextureObject(&costmap_tex_, &resDesc, &texDesc, NULL) );
}

inline void MPPICosts::obstaclesToTexture()
{
    //ST
    //Transfer CPU mem to GPU
    float4* obstacles_ptr = obstacles_costs_.data();
    HANDLE_ERROR(hipMemcpyToArray(obstaclesArray_d_, 0, 0, obstacles_ptr, width_*height_*sizeof(float4), hipMemcpyHostToDevice));
    hipStreamSynchronize(stream_);

    //Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = obstaclesArray_d_;

    //Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;  //add another mode
    texDesc.filterMode = hipFilterModePoint;   //  the returned value is the texel whose texture coordinates are the closest to the input texture coordinates.
    texDesc.readMode = hipReadModeElementType;  // no conversion is performed
    texDesc.normalizedCoords = 1;               // coordinates are normalised (0,1-1/N)

    //Destroy current texture and create new texture object
    HANDLE_ERROR(hipDestroyTextureObject(obstacles_tex_));
    HANDLE_ERROR(hipCreateTextureObject(&obstacles_tex_, &resDesc, &texDesc, NULL) );
}

inline void MPPICosts::costmap3DToTexture()
{
    //ST
    //Transfer CPU mem to GPU
    float4* costmap3D_ptr = costmap3D_costs_.data();
    //ST 3D copying to cuda array works differently
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)costmap3D_ptr, width_*sizeof(float4), width_, height_);
    copyParams.dstArray = costmap3DArray_d_;
    copyParams.extent = make_hipExtent(width_, height_, depth_);;
    copyParams.kind = hipMemcpyHostToDevice;
    HANDLE_ERROR(hipMemcpy3D(&copyParams));
    hipStreamSynchronize(stream_);

    //Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = costmap3DArray_d_;

    //Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;  //add another mode
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;   //  the returned value is the texel whose texture coordinates are the closest to the input texture coordinates.
    texDesc.readMode = hipReadModeElementType;  // no conversion is performed
    texDesc.normalizedCoords = 1;               // coordinates are normalised (0,1-1/N)

    //Destroy current texture and create new texture object
    HANDLE_ERROR(hipDestroyTextureObject(costmap3D_tex_));
    HANDLE_ERROR(hipCreateTextureObject(&costmap3D_tex_, &resDesc, &texDesc, NULL) );
}

inline void MPPICosts::updateParams(ros::NodeHandle nh)
{
  //Transfer to the cost params struct
  l1_cost_ = getRosParam<bool>("l1_cost", nh);
  params_.desired_speed = getRosParam<double>("desired_speed", nh);
  params_.speed_coeff = getRosParam<double>("speed_coefficient", nh);
  params_.track_coeff = getRosParam<double>("track_coefficient", nh);
  params_.max_slip_ang = getRosParam<double>("max_slip_angle", nh);
  params_.slip_penalty = getRosParam<double>("slip_penalty", nh);
  params_.track_slop = getRosParam<double>("track_slop", nh);
  params_.crash_coeff = getRosParam<double>("crash_coeff", nh);
  params_.steering_coeff = getRosParam<double>("steering_coeff", nh);
  params_.throttle_coeff = getRosParam<double>("throttle_coeff", nh);
  params_.boundary_threshold = getRosParam<double>("boundary_threshold", nh);
  params_.discount = getRosParam<double>("discount", nh);
  params_.num_timesteps = getRosParam<int>("num_timesteps", nh);

  //ST
  params_.user_desired_linear_speed=getRosParam<double>("user_desired_linear_speed", nh);
  params_.user_desired_angular_speed=getRosParam<double>("user_desired_angular_speed", nh);
  params_.angular_speed_coeff=getRosParam<double>("angular_speed_coefficient", nh);
  params_.linear_speed_coeff=getRosParam<double>("linear_speed_coefficient", nh);
  params_.smoothness_coeff=getRosParam<double>("smoothness_coefficient", nh);
  params_.goal_coeff=getRosParam<double>("goal_coefficient", nh);
  //end

  //Move the updated parameters to gpu memory
  paramsToDevice();
}

inline void MPPICosts::updateTransform(Eigen::MatrixXf m, Eigen::ArrayXf trs){
  params_.r_c1.x = m(0,0);
  params_.r_c1.y = m(1,0);
  params_.r_c1.z = m(2,0);
  params_.r_c2.x = m(0,1);
  params_.r_c2.y = m(1,1);
  params_.r_c2.z = m(2,1);
  params_.r_c3.x = m(0,2); //ST
  params_.r_c3.y = m(1,2); //ST
  params_.r_c3.z = m(2,2); //ST
  params_.trs.x = trs(0);
  params_.trs.y = trs(1);
  params_.trs.z = trs(2);
  //Move the updated parameters to gpu memory
  paramsToDevice();
}

inline std::vector<float4> MPPICosts::loadTrackData(std::string map_path, Eigen::Matrix3f &R, Eigen::Array3f &trs)
{
  if (!fileExists(map_path)){
    ROS_FATAL("Could not load costmap at path: %s", map_path.c_str());
  }
  cnpy::npz_t map_dict = cnpy::npz_load(map_path);
  float x_min, x_max, y_min, y_max, ppm;
  float z_min, z_max; //ST: heading
  float* xBounds = map_dict["xBounds"].data<float>();
  float* yBounds = map_dict["yBounds"].data<float>();
  float* pixelsPerMeter = map_dict["pixelsPerMeter"].data<float>();
  x_min = xBounds[0];
  x_max = xBounds[1];
  y_min = yBounds[0];
  y_max = yBounds[1];
  //ST: Heading lies between 0 to 360 only. Remember that autorally does not wrap around the heading....
  z_min = 0.0;
  z_max = 2.0*PI;
  ppm = pixelsPerMeter[0];

  width_ = int((x_max - x_min)*ppm);
  height_ = int((y_max - y_min)*ppm);
  depth_ = int(HEADING_BINS);  //ST

  initCostmap();
  initObstacles();
  initCostmap3D(); //ST: just initialise here - updates are done later

  std::vector<float4> track_costs(width_*height_);  //track/ obstacle costs are in 2D
  
  float* channel0 = map_dict["channel0"].data<float>();
  float* channel1 = map_dict["channel1"].data<float>();
  float* channel2 = map_dict["channel2"].data<float>();
  float* channel3 = map_dict["channel3"].data<float>();

  for (int i = 0; i < width_*height_; i++){
    track_costs[i].x = channel0[i];
    track_costs[i].y = channel1[i];
    track_costs[i].z = channel2[i];
    track_costs[i].w = channel3[i];
  }

    //Save the scaling and offset
    //  ST: the Z scaling is done at each conversion since I dont want to break the existing implementation
    R << 1./(x_max - x_min), 0,                  0,
       0,                  1./(y_max - y_min), 0,
       0,                   0,                  1.;
  trs << -x_min/(x_max - x_min), -y_min/(y_max - y_min), 1.;  //not updating translation here because it will hurt backward compatibility
  return track_costs;
}

inline void MPPICosts::paramsToDevice()
{
  HANDLE_ERROR( hipMemcpy(params_d_, &params_, sizeof(CostParams), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipStreamSynchronize(stream_) );
}

inline void MPPICosts::getCostInfo()
{
}

inline float MPPICosts::getDesiredSpeed()
{
  return params_.desired_speed;
}

inline void MPPICosts::setDesiredSpeed(float desired_speed)
{
  params_.desired_speed = desired_speed;
  paramsToDevice();
}

inline void MPPICosts::debugDisplayInit()
{
    debugDisplayInit(10, 10, 50);
    //debugDisplayInit(20, 20, 50);
}

inline void MPPICosts::debugDisplayCostmapInit()
{
    debugDisplayCostmapInit(10, 10, 50);
    //debugDisplayInit(20, 20, 50);
}

inline void MPPICosts::debugDisplayInit(int width_m, int height_m, int ppm)
{
  debug_img_width_ = width_m;
  debug_img_height_ = height_m;
  debug_img_ppm_ = ppm;
  debug_img_size_ = (width_m*ppm)*(height_m*ppm);
  debug_data_ = new float[debug_img_size_];
  debugging_ = true;
  HANDLE_ERROR( hipMalloc((void**)&debug_data_d_, debug_img_size_*sizeof(float)) );
}

//ST: new code to display 3D costmap in 2D
inline void MPPICosts::debugDisplayCostmapInit(int width_m, int height_m, int ppm)
{
    debug_img_costmap_width_ = width_m;
    debug_img_costmap_height_ = height_m;
    debug_img_costmap_ppm_ = ppm;
    debug_img_costmap_size_ = (width_m*ppm)*(height_m*ppm);
    debug_costmap_data_ = new float[debug_img_costmap_size_];
    debugging_costmap_ = true;
    HANDLE_ERROR( hipMalloc((void**)&debug_costmap_data_d_, debug_img_costmap_size_*sizeof(float)) );
}

inline cv::Mat MPPICosts::getDebugDisplay(float x, float y, float heading)
{
  cv::Mat debug_img; ///< OpenCV matrix for display debug info.
  if (!debugging_){
    debugDisplayInit();
  }
  launchDebugCostKernel(x, y, heading, debug_img_width_, debug_img_height_, debug_img_ppm_, 
                        obstacles_tex_, debug_data_d_, params_.r_c1, params_.r_c2, params_.trs, stream_);
  //Now we just have to display debug_data_d_
  HANDLE_ERROR( hipMemcpy(debug_data_, debug_data_d_, debug_img_size_*sizeof(float), hipMemcpyDeviceToHost) );
  hipStreamSynchronize(stream_);
  debug_img = cv::Mat(debug_img_width_*debug_img_ppm_, debug_img_height_*debug_img_ppm_, CV_32F, debug_data_);
  return debug_img;
}

inline cv::Mat MPPICosts::getDebugCostmapDisplay(float x, float y, float heading)
{
    cv::Mat debug_img_costmap; ///< OpenCV matrix for display debug info.
    if (!debugging_costmap_){
        debugDisplayCostmapInit();
    }

    //Uncomment this to enable 2D costmap no 3D
//    launchDebugCostKernel(x, y, heading, debug_img_costmap_width_, debug_img_costmap_height_, debug_img_costmap_ppm_,
//                          costmap_tex_, debug_costmap_data_d_, params_.r_c1, params_.r_c2, params_.trs, stream_);

//wrap the heading in 0-2*PI
    double heading_wrapped = heading;
    heading_wrapped = fmod(heading_wrapped,2.0*PI);
    if (heading_wrapped < 0)
        heading_wrapped += 2.0*PI;

    launchDebugCost3DKernel(x, y, heading_wrapped, debug_img_costmap_width_, debug_img_costmap_height_, debug_img_costmap_ppm_,
                          costmap3D_tex_, debug_costmap_data_d_, params_.r_c1, params_.r_c2, params_.r_c3, params_.trs, stream_);

    //Now we just have to display debug_data_costmap_d_
    HANDLE_ERROR( hipMemcpy(debug_costmap_data_, debug_costmap_data_d_, debug_img_costmap_size_*sizeof(float), hipMemcpyDeviceToHost) );
    hipStreamSynchronize(stream_);
    debug_img_costmap = cv::Mat(debug_img_width_*debug_img_ppm_, debug_img_height_*debug_img_ppm_, CV_32F, debug_costmap_data_);
    return debug_img_costmap;
}

inline void MPPICosts::freeCudaMem()
{
  HANDLE_ERROR(hipDestroyTextureObject(costmap_tex_));
  HANDLE_ERROR(hipDestroyTextureObject(obstacles_tex_));
  HANDLE_ERROR(hipDestroyTextureObject(costmap3D_tex_));
  HANDLE_ERROR(hipFreeArray(obstaclesArray_d_));
  HANDLE_ERROR(hipFreeArray(costmapArray_d_));
  HANDLE_ERROR(hipFreeArray(costmap3DArray_d_));
  HANDLE_ERROR(hipFree(params_d_));
  if (debugging_) {
    HANDLE_ERROR(hipFree(debug_data_d_));
  }
  if(debugging_costmap_){
      HANDLE_ERROR(hipFree(debug_costmap_data_d_));
  }
}

inline void MPPICosts::updateCostmap(std::vector<float> description, std::vector<float> data){
    float *costmap_data = &data[0];
    costmapToTexture(costmap_data);
    HANDLE_ERROR( hipStreamSynchronize(stream_) );

}

inline void MPPICosts::updateGridsCostmap(std::vector<float> description, std::vector<std::vector<float>> data){

    std::vector<float> data_flat(width_*height_*depth_,0.0);
    //Indexing here was the keep to correct 3D costmap building
    for (int z = 0; z < depth_; z++)
        for (int y = 0; y < height_; y++)
            for (int x = 0; x < width_; x++)
                data_flat[z*width_*height_+y*width_+x] = data[z][y*width_+x];
    int offset = 0;
    float *costmap3D_data = &data_flat[0 + offset*(width_*height_)];
    costmap3DToTexture(costmap3D_data);
    HANDLE_ERROR( hipStreamSynchronize(stream_) );
//    costmap3D_data = &data_flat[0 + 1*(width_*height_)];
//    costmapToTexture(costmap3D_data);
//    HANDLE_ERROR( hipStreamSynchronize(stream_) );

}


inline void MPPICosts::updateObstacles(std::vector<float> description, std::vector<float> data){
    float* obs_data = &data[0];
    obstaclesToTexture(obs_data);
    HANDLE_ERROR( hipStreamSynchronize(stream_) );

}




inline __host__ __device__ void MPPICosts::getCrash(float* state, int* crash) {
  if (fabs(state[3]) > 1.57) {
    crash[0] = 1;
  }
}

inline __host__ __device__ float MPPICosts::getControlCost(float* u, float* du, float* vars)
{
  float control_cost = 0;
  control_cost += params_d_->steering_coeff*du[0]*(u[0] - du[0])/(vars[0]*vars[0]);
  control_cost += params_d_->throttle_coeff*du[1]*(u[1] - du[1])/(vars[1]*vars[1]);
  return control_cost;
}

inline __host__ __device__ float MPPICosts::getSpeedCost(float* s, int* crash)
{
  float cost = 0;
  float error = s[4] - params_d_->desired_speed;
  if (l1_cost_){
    cost = fabs(error);
  }
  else {
    cost = error*error;
  }
  return (params_d_->speed_coeff*cost);
}

inline __host__ __device__ float MPPICosts::getCrashCost(float* s, int* crash, int timestep)
{
  float crash_cost = 0;
  if (crash[0] > 0) {
      crash_cost = params_d_->crash_coeff;
  }
  return crash_cost;
}

inline __host__ __device__ float MPPICosts::getStabilizingCost(float* s)
{
  float stabilizing_cost = 0;
  if (fabs(s[4]) > 0.001) {
    float slip = -atan(s[5]/fabs(s[4]));
    stabilizing_cost = params_d_->slip_penalty*powf(slip,2);
    if (fabs(-atan(s[5]/fabs(s[4]))) > params_d_->max_slip_ang) {
      //If the slip angle is above the max slip angle kill the trajectory.
      stabilizing_cost += params_d_->crash_coeff;
    }
  }
  return stabilizing_cost;
}

inline __host__ __device__ void MPPICosts::coorTransform(float x, float y, float* u, float* v, float* w)
{
        //Compute a projective transform of (x, y, 0, 1)
        u[0] = params_d_->r_c1.x*x + params_d_->r_c2.x*y + params_d_->trs.x;
        v[0] = params_d_->r_c1.y*x + params_d_->r_c2.y*y + params_d_->trs.y;
        w[0] = params_d_->r_c1.z*x + params_d_->r_c2.z*y + params_d_->trs.z;
}

inline __host__ __device__ void MPPICosts::coorTransform3D(float x, float y, float z, float* u, float* v, float* s, float* w)
{
    //Compute a projective transform of (x, y, z, 1)
//    u[0] = params_d_->r_c1.x*x + params_d_->r_c2.x*y + params_d_->r_c3.x*z + params_d_->trs.x;
//    v[0] = params_d_->r_c1.y*x + params_d_->r_c2.y*y + params_d_->r_c3.y*z + params_d_->trs.y;
//    s[0] = params_d_->r_c1.z*x + params_d_->r_c2.z*y + params_d_->r_c3.z*z + ;
//    w[0] = 1;
    u[0] = params_d_->r_c1.x*x + params_d_->r_c2.x*y + params_d_->trs.x;
    v[0] = params_d_->r_c1.y*x + params_d_->r_c2.y*y + params_d_->trs.y;
    s[0] = 0;
    w[0] = params_d_->r_c1.z*x + params_d_->r_c2.z*y + params_d_->trs.z;
}

inline __device__ float MPPICosts::getTrackCost(float* s, int* crash)
{
  float track_cost = 0;

  //Compute a transformation to get the (x,y) positions of the front and back of the car.
  float x_front = s[0] + FRONT_D*__cosf(s[2]);
  float y_front = s[1] + FRONT_D*__sinf(s[2]);
  float x_back = s[0] + BACK_D*__cosf(s[2]);
  float y_back = s[1] + BACK_D*__sinf(s[2]);

  float u,v,w; //Transformed coordinates

  //Cost of front of the car
  coorTransform(x_front, y_front, &u, &v, &w);
  float4 track_params_front = tex2D<float4>(obstacles_tex_, u/w, v/w);

  //Cost for back of the car
  coorTransform(x_back, y_back, &u, &v, &w);
  float4 track_params_back = tex2D<float4>(obstacles_tex_, u/w, v/w);

  float track_cost_front = track_params_front.x;
  float track_cost_back = track_params_back.x;

  track_cost = (fabs(track_cost_front) + fabs(track_cost_back) )/2.0;
  if (fabs(track_cost) < params_d_->track_slop) {
    track_cost = 0;
  }
  else {
    track_cost = params_d_->track_coeff*track_cost;
  }
  if (track_cost_front >= params_d_->boundary_threshold || track_cost_back >= params_d_->boundary_threshold) {
    crash[0] = 1;
  }
  return track_cost;
}

inline __device__ float MPPICosts::getDesirabilityCost(float* s, int* crash)
{
    float u,v,w; //Transformed coordinates
    //Cost of front of the car
    coorTransform(s[0], s[1], &u, &v, &w);
    float4 costmap_params_front = tex2D<float4>(costmap_tex_, u/w, v/w);

    float desirability_cost = 1.0 - costmap_params_front.x;
    desirability_cost = params_d_->goal_coeff*desirability_cost;
    return desirability_cost;
}

inline __host__ __device__ float MPPICosts::getGoalCost(float *s)
{
    //float x = 3.0, y = -2.2, yaw = 0.35; // horizontal
   // float x = -1.0, y = 2.0, yaw = 0.75; //vertical
     float x = 0, y = 0, yaw = s[2]; //diagonal

    float dx = s[0] - x;
    float dy = s[1] - y;
    float dyaw = s[2] - yaw;
    float euclid_dist = sqrtf(dx * dx + dy * dy + dyaw * dyaw);
    float goal_cost = params_d_->goal_coeff * euclid_dist;
    return goal_cost;
}

inline __device__ float MPPICosts::getObstacleCost(float* s, int* crash)
{
    float obstacle_cost = 0;
    float fourtyfive_rads = 0.785398;

    //Compute a transformation to get the (x,y) positions of the 4 corners of the car.
    //corner 1 and 3
    float x_front = s[0] + FRONT_DIAGONAL*__cosf(s[2] + fourtyfive_rads);
    float y_front = s[1] + FRONT_DIAGONAL*__sinf(s[2] + fourtyfive_rads);
    float x_back = s[0] + BACK_DIAGONAL*__cosf(s[2] + fourtyfive_rads);
    float y_back = s[1] + BACK_DIAGONAL*__sinf(s[2] + fourtyfive_rads);

    float u,v,w; //Transformed coordinates

    //Cost of front of the car
    coorTransform(x_front, y_front, &u, &v, &w);
    float4 track_params_front = tex2D<float4>(obstacles_tex_, u/w, v/w);

    //Cost for back of the car
    coorTransform(x_back, y_back, &u, &v, &w);
    float4 track_params_back = tex2D<float4>(obstacles_tex_, u/w, v/w);

    float track_cost_front = track_params_front.x;
    float track_cost_back = track_params_back.x;

    obstacle_cost = fmaxf(fabs(track_cost_front), fabs(track_cost_back));

    if (track_cost_front >= params_d_->boundary_threshold || track_cost_back >= params_d_->boundary_threshold) {
        crash[0] = 1;
    }

    //corner 2 and 4 - 135 degree plus heading
    x_front = s[0] + FRONT_DIAGONAL*__cosf(s[2] + 3.0*fourtyfive_rads);
    y_front = s[1] + FRONT_DIAGONAL*__sinf(s[2] + 3.0*fourtyfive_rads);
    x_back = s[0] + BACK_DIAGONAL*__cosf(s[2] + 3.0*fourtyfive_rads);
    y_back = s[1] + BACK_DIAGONAL*__sinf(s[2] + 3.0*fourtyfive_rads);

    //Cost of front of the car
    coorTransform(x_front, y_front, &u, &v, &w);
    track_params_front = tex2D<float4>(obstacles_tex_, u/w, v/w);

    //Cost for back of the car
    coorTransform(x_back, y_back, &u, &v, &w);
    track_params_back = tex2D<float4>(obstacles_tex_, u/w, v/w);

    track_cost_front = track_params_front.x;
    track_cost_back = track_params_back.x;

    obstacle_cost = fmaxf(obstacle_cost, fabs(track_cost_front));
    obstacle_cost = fmaxf(obstacle_cost, fabs(track_cost_back));
    obstacle_cost = params_d_->track_coeff*obstacle_cost;

    if (track_cost_front >= params_d_->boundary_threshold || track_cost_back >= params_d_->boundary_threshold) {
        crash[0] = 1;
    }
    return obstacle_cost;
}


inline __host__ __device__ float MPPICosts::getLinearSpeedCost(float* s, int* crash)
{
    float cost = 0.0;
    float error = s[4] - params_d_->user_desired_linear_speed;
    //float error = sqrt(s[4]*s[4] + s[5]*s[5]) - params_d_->user_desired_linear_speed;
    if (l1_cost_){
        cost = fabs(error);
    }
    else {
        cost = error*error;
    }
    return (params_d_->linear_speed_coeff*cost);
}

inline __host__ __device__ float MPPICosts::getAngularVelocityCost(float* s, int* crash)
{
    float cost = 0;
    float yaw_velocity_ = s[6];
    float error = yaw_velocity_ - params_d_->user_desired_angular_speed;
    if (l1_cost_){
        cost = fabs(error);
    }
    else {
        cost = error*error;
    }
    return params_d_->angular_speed_coeff*cost;
}


inline __host__ __device__ float MPPICosts::getStopCall(float* s, int* crash)
{
    float cost = 0.0;
    if(fabs(params_d_->user_desired_linear_speed)  <= 0.0001 && fabs(params_d_->user_desired_angular_speed) <= 0.0001)
        cost = params_d_->crash_coeff;
    return cost;
}

inline __host__ __device__ float MPPICosts::getSmoothnessCost(float* u, float* prev_du)
{
    float cost = 0.0;
    cost += params_d_->smoothness_coeff*fabs(u[0] - prev_du[0]); // throttle
    cost += params_d_->smoothness_coeff*fabs(u[1] - prev_du[1]);    //steering
    return cost;
}

inline __device__ float MPPICosts::getCostmap3DCost(float* s)
{
    float desirability_cost = 0;
    float x = s[0], y = s[1], heading = s[3];
    float u,v,w; //Transformed coordinates
    //Cost of front of the
    coorTransform(s[0], s[1], &u, &v, &w);
    double heading_wrapped = heading;
    heading_wrapped = fmod(heading_wrapped,2.0*PI);
    if (heading_wrapped < 0)
        heading_wrapped += 2.0*PI;
    //scale the heading to 0-1 here
    float theta = heading_wrapped/(2.0*PI);
    float4 costmap_params_front = tex3D<float4>(costmap3D_tex_, u/w, v/w, theta);
    desirability_cost = 1.0 - costmap_params_front.x; //we need the inverse to minimise the cost
    desirability_cost = params_d_->goal_coeff*desirability_cost;
    return desirability_cost;
}

//Compute the immediate running cost.
inline __device__ float MPPICosts::computeCost(float* s, float* u, float* du, 
                                        float* vars, int* crash, int timestep, float* prev_du)
{
//    if(timestep == 9)
//        printf("%f - %f\n", params_d_->user_desired_angular_speed, s[6]);

    //obstacle cost
    float track_cost = getTrackCost(s, crash);
    float crash_cost = powf(params_.discount, timestep)*getCrashCost(s, crash, timestep);

    //costmap cost
    float desirability_3D_cost = 0.0;//getCostmap3DCost(s);
    float desirability_2D_cost = 0.0;//getDesirabilityCost(s,crash);

    //control cost
    float control_cost = 0.0;//getControlCost(u, du, vars);

    // jerk cost or smoothness cost
    float smoothness_cost = 0.0;//getSmoothnessCost(u,prev_du);

    // speed costs - linear + angular + stop call
    float linear_velocity_cost = powf(params_.discount, timestep)*getLinearSpeedCost(s, crash); // time decay left
    float angular_velocity_cost = exp(-timestep/25.0)*getAngularVelocityCost(s, crash); //time decay left
    float user_joystick_cost = linear_velocity_cost + angular_velocity_cost;



    //obsolete functions
    float speed_cost = 0;//getSpeedCost(s, crash);
    float goal_cost = 0.0;//getGoalCost(s);
    float obstacle_cost = 0;//getObstacleCost(s,crash);
    float stabilizing_cost = getStabilizingCost(s); //slip angle
    float stop_call_cost = 0;//getStopCall(s,crash); //stops the robot to move if the joystick is stopped

    float cost = control_cost + crash_cost + track_cost + smoothness_cost +
            user_joystick_cost + desirability_2D_cost + desirability_3D_cost + goal_cost;

  if (cost > 1e12 || isnan(cost)) {
    cost = 1e12;
  }
  return cost;
}

inline __device__ float MPPICosts::terminalCost(float* s)
{
  return 0.0;
}

}


